#include "hip/hip_runtime.h"
#include "wb.h"
#include<bits/stdc++.h>
using namespace std;

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define mask_width 5
#define mask_radius mask_width / 2
#define TILE_WIDTH 12
#define w (TILE_WIDTH + mask_width - 1)
#define clamp(x) (min(max((x), 0.0), 1.0))



unsigned char * getArrayFromPpm(const char * filename,int *imageHeight,int *imageWidth){
  cout<<"Reading file : "<<filename<<"\n";
  string s;
  ifstream inFile(filename);
  getline(inFile,s);
  // cout<<s<<"\n";
  getline(inFile,s);
  // cout<<s<<"\n";
  inFile >> *imageWidth;
  // cout<<*imageWidth<<"\n";
  inFile >> *imageHeight;
  // cout<<*imageHeight<<"\n";
  int t = 0;
  inFile >> t;
  // cout<<t<<"\n";
  int temp;
  unsigned char * conts = (unsigned char *) malloc((*imageHeight) * (*imageWidth) * 3 * sizeof(unsigned char));
  int i=0;
  while(inFile >> temp){
    conts[i++] = temp;
  }
  cout<<"Read finished\n";
  inFile.close();
  return conts; 
}
float * getArrayFromMask(const char * filename){
  cout<<"Reading file : "<<filename<<"\n";
  ifstream inFile(filename);
  float temp;
  float * conts = (float *) malloc(5 * 5 * sizeof(float));
  int i=0;
  while(inFile>>temp){
  	// inFile.read((char*)&temp, sizeof(float));
    // cout<<temp<<" temp\n";
    conts[i++] = temp;
  }

  cout<<"Read finished\n";
  inFile.close();
  return conts; 
}

void wbSolution(const wbArg_t& args, const unsigned char* image)
{
	int imageWidth,imageHeight;
    unsigned char * solnImage = getArrayFromPpm(wbArg_getInputFile(args,0),&imageHeight,&imageWidth);
        cout<<imageHeight<<" "<<imageWidth<<"\n";
        int errCnt = 0;

        for (int i = 0; i < imageHeight*imageWidth*3; ++i)
        {
                    // cout<<solnImage[i]<<" : "<<image[i]<<"\n";
        			// const float error = fabs(solnImage[i] - image[i]);
                    if (solnImage[i] != image[i])
                    {
                        if (errCnt < wbInternal::kErrorReportLimit)
                            std::cout << "Image pixels do not match at position (" << i << "). [" << (int)image[i] << ", " <<  (int)solnImage[i] << "]\n";
                        ++errCnt;
                    }
        }

        if (!errCnt)
            std::cout << "Solution is correct." << std::endl;
        else
            std::cout << errCnt << " tests failed!" << std::endl;
    // }

    // wbImage_delete(solnImage);
}

//@@ INSERT CODE HERE


// unsigned char * compute(unsigned char *data, float *mask, int height,
//              int width) {

//   const int num_channels = 3;

//   float inputData[height * width * num_channels];
//   for(int i =0 ;i<height*width*num_channels;++i){
//       inputData[i] = ((int)data[i])/255.0;
//   }

//   float *outputData = (float *) malloc(height*width*3*sizeof(float));

//   int img_width  = width;
//   int img_height = height;
//   int mask_rows = 5;
//   int mask_cols = 5;
//   int mask_radius_y = mask_rows / 2; // 5 X 5 mask matrix is fixed
//   int mask_radius_x = mask_cols / 2;
//   for (int out_y = 0; out_y < img_height; ++out_y) {
//     for (int out_x = 0; out_x < img_width; ++out_x) {
//       for (int c = 0; c < num_channels; ++c) { // channels
//         float acc = 0;
//         for (int off_y = -mask_radius_y; off_y <= mask_radius_y; ++off_y) {
//           for (int off_x = -mask_radius_x; off_x <= mask_radius_x;
//                ++off_x) {
//             int in_y   = out_y + off_y;
//             int in_x   = out_x + off_x;
//             int mask_y = mask_radius_y + off_y;
//             int mask_x = mask_radius_x + off_x;
//             if (in_y < img_height && in_y >= 0 && in_x < img_width &&
//                 in_x >= 0) {
//               acc +=
//                   (inputData[(in_y * img_width + in_x) * num_channels + c]) *
//                   mask[mask_y * mask_cols + mask_x];
//                   // cout<<mask[mask_y * mask_cols + mask_x]<<"\n";
//             } else {
//               acc += 0.0f;
//             }
//           }
//         }
//         // fprintf(stderr, "%f %f\n", clamp(acc));
//         cout<<"acc : "<<acc<<"\n";
//         outputData[(out_y * img_width + out_x) * num_channels + c] =
//             clamp(acc);
//       }
//     }
//   }
//   unsigned char *output = (unsigned char *) malloc(height*width*3*sizeof(unsigned char));
//   for(int i =0;i<height*width*num_channels;++i){
//       output[i] = (unsigned char) floor(outputData[i] * 255);
//       // cout<<i<<" helloo "<<outputData[i]<<"\n";
//   }
//   return output;
// }

static void write_data(const char *file_name, unsigned char *data,
                       unsigned int width, unsigned int height,
                       unsigned int channels) {
    FILE *handle = fopen(file_name, "w");
    fprintf(handle, "P6\n");
    fprintf(handle, "#Created by %s\n", __FILE__);
    fprintf(handle, "%d %d\n", width, height);
    fprintf(handle, "255\n");
    for(int i=0;i<width*height*channels;++i){
    	fprintf(handle,"%d ",data[i]);
    }
  fflush(handle);
  fclose(handle);
}


// // using global memory kernel

// __global__ 
// void convolution(float * deviceInputImageData, float *deviceMaskData,unsigned char *deviceOutputImageData,int imageChannels,int imageWidth,int imageHeight){
// 	int y = blockIdx.y*blockDim.y + threadIdx.y;
// 	int x = blockIdx.x*blockDim.x + threadIdx.x;

//     if(y < imageHeight && x < imageWidth){

//         	float acc = 0;
// 	        for (int off_y = -mask_radius; off_y <= mask_radius; ++off_y) {
// 	          for (int off_x = -mask_radius; off_x <= mask_radius;++off_x) {
// 	            int in_y   = y + off_y;
// 	            int in_x   = x + off_x;
// 	            int mask_y = mask_radius + off_y;
// 	            int mask_x = mask_radius + off_x;
// 	            if (in_y < imageHeight && in_y >= 0 && in_x < imageWidth && in_x >= 0) {
// 	              acc += deviceInputImageData[(in_y * imageWidth + in_x)] * deviceMaskData[mask_y * mask_width + mask_x];
// 	            } else {
// 	              acc += 0.0f;
// 	            }
// 	          }
// 	        }
// 	        // deviceOutputImageData[(y * imageWidth + x) * imageChannels + c] = (deviceInputImageData[(y * imageWidth + x) * imageChannels + c]);    

// 	        deviceOutputImageData[(y * imageWidth + x)] = (unsigned char)(floor(clamp(acc)*255));    
//     }
// }

__global__ 
void convolution(float * deviceInputImageData, const float * __restrict__ deviceMaskData,unsigned char *deviceOutputImageData,int imageChannels,int imageWidth,int imageHeight){
	int y = blockIdx.y * TILE_WIDTH + threadIdx.y - mask_radius;
	int x = blockIdx.x * TILE_WIDTH + threadIdx.x - mask_radius;																	

	__shared__ float ds_Image[16][16];
	
	ds_Image[threadIdx.y][threadIdx.x] = 0;

	if(y>=0 && y< imageHeight && x>=0 && x < imageWidth)
	{
		ds_Image[threadIdx.y][threadIdx.x] = deviceInputImageData[y* imageWidth + x];

		__syncthreads();

       	float acc = 0;

       	if(threadIdx.x >= mask_radius && threadIdx.x < 16 - mask_radius && threadIdx.y >= mask_radius
       	 	&& threadIdx.y < 16 - mask_radius){

			    for (int off_y = -mask_radius; off_y <= mask_radius; ++off_y) 
			    {
			        for (int off_x = -mask_radius; off_x <= mask_radius;++off_x) 
			        {
			            int in_y   = threadIdx.y + off_y;
			            int in_x   = threadIdx.x + off_x;
			            int mask_y = mask_radius + off_y;
			            int mask_x = mask_radius + off_x;
			            if (in_y < 16 && in_y >= 0 && in_x < 16 && in_x >= 0) 
			            {
			              acc += ds_Image[in_y][in_x] * deviceMaskData[mask_y * mask_width + mask_x];
			            } 
		            	 else 
		            	{
		              		acc += 0.0f;
		            	}
			          }
			    }

			    // __syncthreads();
			        // deviceOutputImageData[(y * imageWidth + x) * imageChannels + c] = (deviceInputImageData[(y * imageWidth + x) * imageChannels + c]);    

			    deviceOutputImageData[(y * imageWidth + x)] = (unsigned char)(floor(clamp(acc)*255.0));    
	    }
    }
}

__global__
void split(float *ip, float *red_channel, float *green_channel,float *blue_channel, int width, int height)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int row = by*blockDim.y + ty;
	int col = bx*blockDim.x + tx;


	if(row<height && col<width)
	{
		red_channel[row*width + col] = ip[3*(row*width+col)];
		green_channel[row*width + col] = ip[3*(row*width+col) + 1];
		blue_channel[row*width + col] = ip[3*(row*width+col) + 2];
	}

}

__global__
void mergeColors(unsigned char *red_channel, unsigned char *green_channel,unsigned char *blue_channel, unsigned char *output, int width, int height)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int row = by*blockDim.y + ty;
	int col = bx*blockDim.x + tx;


	if(row<height && col<width)
	{
		output[3*(row*width + col)] = red_channel[row*width + col];
		output[3*(row*width + col)+1] = green_channel[row*width + col];
		output[3*(row*width + col)+2] = blue_channel[row*width + col];
	} 

}


int main(int argc, char *argv[]) {
  wbArg_t arg;
  int maskRows;
  int maskColumns;
  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  char *inputMaskFile;
  // wbImage_t inputImage;
  // wbImage_t outputImage;
  unsigned char *tempInputImage;
  float *hostInputImageData;
  unsigned char *hostOutputImageData;
  float *hostMaskData;
  float *deviceInputImageData;
  unsigned char *deviceOutputImageData;
  float *deviceMaskData;

  float *deviceInputRed;
  float *deviceInputGreen;
  float *deviceInputBlue;

  unsigned char *deviceOutputRed;
  unsigned char *deviceOutputGreen;
  unsigned char *deviceOutputBlue;


  arg = wbArg_read(argc, argv); /* parse the input arguments */


  inputImageFile = wbArg_getInputFile(arg, 1);
  inputMaskFile  = wbArg_getInputFile(arg, 2);

  hostMaskData = getArrayFromMask(inputMaskFile);
  // cout<<"MAsk : \n";
  // for(int i =0;i<5;++i){
  // 	for(int j=0;j<5;++j){
  // 		cout<<hostMaskData[i*5+j]<<" ";
  // 	}
  // 	cout<<"\n";
  // }
  tempInputImage =  getArrayFromPpm(inputImageFile,&imageHeight,&imageWidth);
  // cout<<(int)tempInputImage[0]<<"\n";
  imageChannels = 3;
  

  int imageSize = imageHeight*imageWidth*imageChannels;
  hostInputImageData = (float*) malloc(imageSize*sizeof(float));

  for(int i=0;i<imageWidth*imageHeight*imageChannels;++i){
  	hostInputImageData[i] = ((int)tempInputImage[i])/255.0;
  }

  hostOutputImageData = (unsigned char *) malloc(imageSize*sizeof(unsigned char));
  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");


  wbTime_start(GPU, "Doing GPU memory allocation");
  wbCheck(hipMalloc((void **)&deviceInputImageData,imageSize*sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceMaskData,5*5*sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceOutputImageData,imageSize*sizeof(unsigned char)));

  wbCheck(hipMalloc((void **)&deviceInputRed,imageHeight*imageWidth*sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceInputGreen,imageHeight*imageWidth*sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceInputBlue,imageHeight*imageWidth*sizeof(float)));

  wbCheck(hipMalloc((void **)&deviceOutputRed,imageHeight*imageWidth*sizeof(unsigned char)));
  wbCheck(hipMalloc((void **)&deviceOutputGreen,imageHeight*imageWidth*sizeof(unsigned char)));
  wbCheck(hipMalloc((void **)&deviceOutputBlue,imageHeight*imageWidth*sizeof(unsigned char)));



  hipMemset(deviceOutputImageData,0,imageSize*sizeof(unsigned char));
  // //@@ INSERT CODE HERE
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  // //@@ INSERT CODE HERE
  wbCheck(hipMemcpy(deviceInputImageData,hostInputImageData,imageSize*sizeof(float),hipMemcpyHostToDevice));
  // wbCheck(hipMemcpy(deviceOutputImageData,hostOutputImageData,imageSize*sizeof(float),hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceMaskData,hostMaskData,5*5*sizeof(float),hipMemcpyHostToDevice));
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  dim3 dimBlock(16,16,1);
  int gridsx = (imageWidth-1)/TILE_WIDTH + 1;
  int gridsy = (imageHeight-1)/TILE_WIDTH + 1;

  // dim3 dimGrid(2,2,1);
  dim3 dimGrid(gridsx,gridsy,1);
  
  // cout<<hostInputImageData<<" : "<<

  split<<<dimGrid,dimBlock>>>(deviceInputImageData,deviceInputRed,deviceInputGreen,
  								deviceInputBlue,imageWidth,imageHeight);

  convolution<<<dimGrid,dimBlock>>>(deviceInputRed, deviceMaskData,
                                     deviceOutputRed, imageChannels,
                                     imageWidth, imageHeight);
  convolution<<<dimGrid,dimBlock>>>(deviceInputGreen, deviceMaskData,
                                     deviceOutputGreen, imageChannels,
                                     imageWidth, imageHeight);
  convolution<<<dimGrid,dimBlock>>>(deviceInputBlue, deviceMaskData,
                                     deviceOutputBlue, imageChannels,
                                     imageWidth, imageHeight);

  mergeColors<<<dimGrid,dimBlock>>>(deviceOutputRed,deviceOutputGreen,deviceOutputBlue,
  									 deviceOutputImageData,imageWidth,imageHeight);

  // hostOutputImageData =  compute(tempInputImage,hostMaskData,imageHeight,imageWidth);
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  // //@@ INSERT CODE HERE
  wbCheck(hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(unsigned char),
             hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  // cout<<hostInputImageData[0]<<" : "<<(int)hostOutputImageData[0]<<"\n";

  wbSolution(arg, hostOutputImageData); //changed def in wb.h

  // //@@ Insert code here

  // free(hostMaskData);
  // wbImage_delete(outputImage);
  // wbImage_delete(inputImage);

  return 0;
}
