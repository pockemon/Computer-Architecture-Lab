#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<bits/stdc++.h>
#include "wb.h"

#define NUM_BINS 4096

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

__global__ void hist(unsigned int* input, unsigned int* bins, int length)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i < length)
{
    atomicAdd(&bins[input[i]], 1);
  }
}

int main(int argc, char *argv[]) {

  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *deviceInput;
  unsigned int *deviceBins;
  wbArg_t args = wbArg_read(argc, argv);
         
  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (unsigned int *)wbImport(wbArg_getInputFile(args, 1),
                                       &inputLength);
  hostBins = (unsigned int *)calloc(NUM_BINS , sizeof(unsigned int));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);
  wbLog(TRACE, "The number of bins is ", NUM_BINS);

  std::cout<<std::endl;
  for(int i=0;i<10;i++)
    std::cout<<hostInput[i]<<" ";
    std::cout<<std::endl;
wbTime_start(GPU, "Allocating GPU memory.");
hipMalloc((void **)&deviceInput,
             inputLength * sizeof(int));
  hipMalloc((void **)&deviceBins,
             NUM_BINS * sizeof(int));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating GPU memory.");
wbTime_start(GPU, "Copying input memory to the GPU.");
hipMemcpy(deviceInput, hostInput,
             inputLength * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(deviceBins, hostBins,
             NUM_BINS * sizeof(int),
             hipMemcpyHostToDevice);
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Copying input memory to the GPU.");
 wbLog(TRACE, "Launching kernel");


  wbTime_start(Compute, "Performing CUDA computation");
hist<<<(inputLength-1)/128 + 1, 128>>> (deviceInput, deviceBins, inputLength);
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
 hipMemcpy(hostBins, deviceBins, NUM_BINS*sizeof(int), hipMemcpyDeviceToHost);
wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
hipFree(deviceBins);
  hipFree(deviceInput);
  wbTime_stop(GPU, "Freeing GPU Memory");
wbSolution(args, hostBins, NUM_BINS);


  free(hostBins);
  free(hostInput);
  return 0;
}
                                                                                                                            88,1          Bot
